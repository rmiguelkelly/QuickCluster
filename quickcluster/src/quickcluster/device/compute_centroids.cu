#include "hip/hip_runtime.h"

#include <quickcluster/device/common.h>
#include <cstdlib>
#include <cmath>

inline float eclidean_distance(float *v1, float *v2, size_t N) {

    float sum = 0.0;

    for (size_t i = 0; i < N; i++) {
        sum += powf(v1[i] - v2[i], 2);
    }

    return sqrtf(sum);
}

__device__ void compute_nearest_centroids(const float *data, const float *centroids, DataContext *context, unsigned long *results) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;


}