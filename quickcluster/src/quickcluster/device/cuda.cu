
#include <quickcluster/device/hip/hip_runtime.h>

int cuda_find_device(struct gpu_device *device) {

    int cuda_device;
    
    // CUDA result stuff
    hipError_t res;

    // Get the device
    res = hipGetDevice(&cuda_device);

    if (res != hipError_t::hipSuccess) {
        return -1;
    }

    // Get the device properties
    hipDeviceProp_t props;
    res = hipGetDeviceProperties(&props, cuda_device);

    if (res != hipError_t::hipSuccess) {
        return -1;
    }

    device->id = cuda_device;

    int len = strlen(props.name);
    strncpy(device->name, props.name, len);

    return 0;
}


int cuda_device_init(DeviceHandle *handle) {

    // Used to signify that CUDA is to be used
    cuda_device* hndl = new cuda_device();
    *handle = hndl;

    return 0;
}


int cuda_compute_nearest_centroids(DeviceHandle *handle, const float *data, const float *centroids, const DataContext *context, unsigned long *results) {

    return 0;
}


int cuda_device_release(DeviceHandle *handle) {

    // Delete the pointer
    cuda_device *hndl = (cuda_device*)(*handle);
    delete hndl;

    return 0;
}